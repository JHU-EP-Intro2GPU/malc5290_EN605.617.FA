#include "driver/cutlass_driver.hpp"
#include "driver/cutlass_helper.h"
#include "driver/performance_helper.hpp"
#include "matrix/matrix_helper.hpp"

#include "cutlass/gemm/device/gemm.h"

#include <iostream>

using ColumnMajor = cutlass::layout::ColumnMajor;
using IntGemm =
    cutlass::gemm::device::Gemm< int, ColumnMajor, int, ColumnMajor, int, ColumnMajor>;
using ShortGemm =
    cutlass::gemm::device::Gemm< short, ColumnMajor, short, ColumnMajor, short, ColumnMajor>;
using DoubleGemm =
    cutlass::gemm::device::Gemm< double, ColumnMajor, double, ColumnMajor, double, ColumnMajor>;
using FloatGemm =
    cutlass::gemm::device::Gemm< float, ColumnMajor, float, ColumnMajor, float, ColumnMajor>;

template <class T>
void init_data(cutlass::Status& status, const Matrix<T>& mat_a, const Matrix<T>& mat_b, T*& a, T*& b, T*& results, T*& cutlass_a, T*& cutlass_b, T*& cutlass_results )
{    
    hipError_t cuda_result;
    a = (T*)malloc(mat_a.size() * sizeof(T));
    b = (T*)malloc(mat_b.size() * sizeof(T));
    results = (T*)malloc(mat_a.m_size() * mat_b.n_size() * sizeof(T));
    MatrixHelper::copy_data<T>(mat_a, a);
    MatrixHelper::copy_data<T>(mat_b, b);
    int result_m = mat_a.m_size();
    int result_n = mat_b.n_size();

    cuda_result = hipMalloc((void**)&cutlass_a, mat_a.size() * sizeof(T));
    if ( cuda_result != hipSuccess )
    {
        std::cerr << "Failed to allocate matrix a: \n" << hipGetErrorString(cuda_result) << std::endl;
        exit(EXIT_FAILURE);
    }
    
    cuda_result = hipMalloc((void**)&cutlass_b, mat_b.size() * sizeof(T));
    if ( cuda_result != hipSuccess )
    {
        std::cerr << "Failed to allocate matrix b: \n" << hipGetErrorString(cuda_result) << std::endl;
        exit(EXIT_FAILURE);
    }
    
    cuda_result = hipMalloc((void**)&cutlass_results, result_m * result_n * sizeof(T));
    if ( cuda_result != hipSuccess )
    {
        std::cerr << "Failed to allocate matrix cutlass_cuda_results: \n" << hipGetErrorString(cuda_result) << std::endl;
        exit(EXIT_FAILURE);
    }

    cuda_result = hipMemcpy( cutlass_a, a, mat_a.size() * sizeof(T), hipMemcpyHostToDevice);
    if ( cuda_result != hipSuccess )
    {
        std::cerr << "Failed to copy mat a: \n" << hipGetErrorString(cuda_result) << std::endl;
        exit(EXIT_FAILURE);
    }

    cuda_result = hipMemcpy( cutlass_b, b, mat_b.size() * sizeof(T), hipMemcpyHostToDevice); 
    if ( cuda_result != hipSuccess )
    {
        std::cerr << "Failed to copy mat b: \n" << hipGetErrorString(cuda_result) << std::endl;
        exit(EXIT_FAILURE);
    }


//    std::cout << "MATRIX A: \n";
//    MatrixHelper::print_matrix<float>(Orientation::COLUMN_MAJOR, mat_a.m_size(), mat_a.n_size(), (float*)a);
//    std::cout << "MATRIX B: \n";
//    MatrixHelper::print_matrix<float>(Orientation::COLUMN_MAJOR, mat_b.m_size(), mat_b.n_size(), (float*)b);
}

template <class T>
void free_data(T*& a, T*& b, T*& results, T*& cutlass_a, T*& cutlass_b, T*& cutlass_results )
{    
    free(a);
    free(b);
    free(results);
    hipFree(cutlass_a);
    hipFree(cutlass_b);
    hipFree(cutlass_results);
}

template <class T>
cutlass::Status  multiply(T* a, T* b, T* results, T* cutlass_a, T* cutlass_b, T* cutlass_results, const int& result_m, const int& result_n, const int& aNSize )
{
    return cutlass::Status::kInvalid;
}

template <>
cutlass::Status multiply<double>(double* a, double* b, double* results, double* cutlass_a, double* cutlass_b, double* cutlass_results, const int& result_m, const int& result_n, const int& aNSize )
{
    DoubleGemm gemm_operator; 
    
    DoubleGemm::Arguments args({ result_m, result_n, aNSize },
                                {cutlass_a, result_m},
                                {cutlass_b, aNSize},
                                {cutlass_results, result_m},
                                {cutlass_results, result_m},
                                {1, 0});
    return gemm_operator(args);
}

template <>
cutlass::Status multiply<float>(float* a, float* b, float* results, float* cutlass_a, float* cutlass_b, float* cutlass_results, const int& result_m, const int& result_n , const int& aNSize )
{
    FloatGemm gemm_operator; 
    
    FloatGemm::Arguments args({ result_m, result_n, aNSize },
                                {cutlass_a, result_m},
                                {cutlass_b, aNSize},
                                {cutlass_results, result_m},
                                {cutlass_results, result_m},
                                {1, 0});
    return gemm_operator(args);
}

template <>
cutlass::Status multiply<int>(int* a, int* b, int* results, int* cutlass_a, int* cutlass_b, int* cutlass_results, const int& result_m, const int& result_n, const int& aNSize  )
{
    IntGemm gemm_operator; 
    
    IntGemm::Arguments args({ result_m, result_n, aNSize },
                                {cutlass_a, result_m},
                                {cutlass_b, aNSize},
                                {cutlass_results, result_m},
                                {cutlass_results, result_m},
                                {1, 0});
    return gemm_operator(args);
}

template <>
cutlass::Status multiply<short>(short* a, short* b, short* results, short* cutlass_a, short* cutlass_b, short* cutlass_results, const int& result_m, const int& result_n, const int& aNSize  )
{
    ShortGemm gemm_operator; 
    
    ShortGemm::Arguments args({ result_m, result_n, aNSize },
                                {cutlass_a, result_m},
                                {cutlass_b, aNSize},
                                {cutlass_results, result_m},
                                {cutlass_results, result_m},
                                {1, 0});
    return gemm_operator(args);
}

template <class T>
void multiply( const Matrix<T> mat_a, const Matrix<T> mat_b )
{
    cutlass::Status status;

    T* a; 
    T* b;
    T* results;
    T* cutlass_a;
    T* cutlass_b;
    T* cutlass_results;

    int result_m = mat_a.m_size();
    int result_n = mat_b.n_size();
    auto start = get_clock_time();
    init_data<T>(
            status,                // Cublas Status
            mat_a,                 // Matrix A
            mat_b,                 // Matrix B
            a,                     // Pointer to matrix A values
            b,                     // Pointer to matrix B values
            results,               // Pointer to multiplication results
            cutlass_a,              // Device pointer to matrix A values 
            cutlass_b,              // Device pointer to matrix B values
            cutlass_results);       // Device pointer to multiplication results
    auto stop = get_clock_time();
    std::cout << get_duration_seconds(start, stop) << " ";

#ifdef DEBUG
    std::cout << "A: M" << mat_a.m_size() << "\n";
    std::cout << "A: N" << mat_a.n_size() << "\n";
    std::cout << "B: M" << mat_b.m_size() << "\n";
    std::cout << "B: N" << mat_b.n_size() << "\n";
#endif
    start = get_clock_time();
    status = multiply(a, b, results, cutlass_a, cutlass_b, cutlass_results, result_m, result_n, mat_a.n_size());
    hipDeviceSynchronize();
    stop = get_clock_time();
    if (status != cutlass::Status::kSuccess) {
        free_data<T>(
            a,
            b,
            results,
            cutlass_a,
            cutlass_b,
            cutlass_results);   
        std::cerr << "FAILED TO RUN GEMM\n";
        exit(EXIT_FAILURE);
    }
    std::cout << get_duration_seconds(start, stop) << " ";

    start = get_clock_time();
    hipError_t cuda_result = hipMemcpy( results, cutlass_results, result_m * result_n * sizeof(T), hipMemcpyDeviceToHost);
    stop = get_clock_time();
    std::cout << get_duration_seconds(start, stop) << "\n";
    if ( cuda_result != hipSuccess )
    {
        std::cerr << "Failed to copy result post gemm: \n" << hipGetErrorString(cuda_result) << std::endl;
        exit(EXIT_FAILURE);
    }

    //MatrixHelper::print_matrix<short>(Orientation::ROW_MAJOR, result_m, result_n, results);
    //MatrixHelper::print_matrix<T>(Orientation::COLUMN_MAJOR, result_m, result_n, results);

    free_data<T>(
            a,
            b,
            results,
            cutlass_a,
            cutlass_b,
            cutlass_results);   

}

template <class T>
void CutlassDriver<T>::multiply_matrices()
{
//    std::cout << "CUTLASS DRIVER: MULTIPLY_MATRICES\n";
    if ( MatrixDriver<T>::_mat_a.orientation() == Orientation::ROW_MAJOR )
    {
        MatrixHelper::change_orientation(MatrixDriver<T>::_mat_a,Orientation::COLUMN_MAJOR); 
    }
    if ( MatrixDriver<T>::_mat_b.orientation() == Orientation::ROW_MAJOR )
    {
        MatrixHelper::change_orientation(MatrixDriver<T>::_mat_b,Orientation::COLUMN_MAJOR); 
    }
    //std::cout << "ORIENTATION CHANGED TO COLUMN MAJOR FOR CUTLASS\n";
    multiply<T>(MatrixDriver<T>::_mat_a, MatrixDriver<T>::_mat_b);
}

template class CutlassDriver<int>;
template class CutlassDriver<short>;
template class CutlassDriver<double>;
template class CutlassDriver<float>;
