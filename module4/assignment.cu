//Based on the work of Andrew Krepps

#include <hip/hip_runtime.h>
#include <chrono>
#include <fstream>
#include <random>
#include <stdio.h>
#include <string>
#include <iostream>

typedef struct {
    unsigned int a;
    unsigned int b;
} MathStruct; 

typedef struct {
    int add;
    int sub;
    int mult;
    int mod;
    int cipher;
} ResultsStruct;
// Uses the GPU to add the block + thread index in array_a to array_b to array_results
__global__
void add_arrays( 
    const MathStruct* const data,
    ResultsStruct* const results)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    results[index].add = data[index].a + data[index].b;
}

// Uses the GPU to subtract the block + thread index in array_b from array_a to array_results
__global__
void sub_arrays( 
    const MathStruct* const data,
    ResultsStruct* const results)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    results[index].sub = data[index].a - data[index].b;
}


// Uses the GPU to multiply the block + thread index in array_a by array_b to array_results
__global__
void mult_arrays( 
    const MathStruct* const data,
    ResultsStruct* const results)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    results[index].mult = data[index].a * data[index].b;
}


// Uses the GPU to mudulot the block + thread index in array_a by array_b to array_results
__global__
void mod_arrays( 
    const MathStruct* const data,
    ResultsStruct* const results)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    results[index].mod = data[index].a % data[index].b;
}

__host__
void print_results(
    const std::string& outputName, const int& totalThreads, const int& blockSize, const int& add_time,
    const int& sub_time, const int& mult_time, const int& mod_time, const MathStruct* const  data,
    const ResultsStruct* const results)
{
    std::ofstream stream(outputName);
    if (stream.is_open())
    {
        stream << "Results with Thread Count: " << totalThreads << " and Block Size: " << blockSize << "\n";
        stream << "Add Time nanoseconds:\t" << add_time << "\n";
        stream << "Sub Time nanoseconds:\t" << sub_time << "\n";
        stream << "Mult Time nanoseconds:\t" << mult_time << "\n";
        stream << "Mod Time nanoseconds:\t" << mod_time << "\n";

        stream << "Add Results:\n";
        for( int i = 0; i < totalThreads; i++ )
        {
            stream << "A(" << data[i].a << ") + B("  << data[i].b << ") = " <<  results[i].add << "\n";
        }
        
        stream << "\n\nSub Results:\n";
        for( int i = 0; i < totalThreads; i++ )
        {
            stream << "A(" << data[i].a << ") - B("  << data[i].b << ") = " <<  results[i].sub << "\n";
        }
        
        stream << "\n\nMult Results:\n";
        for( int i = 0; i < totalThreads; i++ )
        {
            stream << "A(" << data[i].a << ") * B("  << data[i].b << ") = " <<  results[i].mult << "\n";
        }
        
        stream << "\n\nMult Results:\n";
        for( int i = 0; i < totalThreads; i++ )
        {
            stream << "A(" << data[i].a << ") % B("  << data[i].b << ") = " <<  results[i].mod << "\n";
        }
    
    }
    else{
        printf("FILE NOT OPEN?\n");
    }
    stream.close();
}

__host__
void run_kernal(
    const int& blockSize, const int& totalThreads, const int& numBlocks, const std::string& outputName,
    const MathStruct* const data, ResultsStruct*& results, MathStruct* d_data, ResultsStruct* d_results)
{
    auto start = std::chrono::high_resolution_clock::now();
    add_arrays<<<numBlocks, blockSize>>>(d_data, d_results);
    auto stop = std::chrono::high_resolution_clock::now();
    auto add_time = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();

    start = std::chrono::high_resolution_clock::now();
    sub_arrays<<<numBlocks, blockSize>>>(d_data, d_results);
    stop = std::chrono::high_resolution_clock::now();
    auto sub_time = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
    
    start = std::chrono::high_resolution_clock::now();
    mult_arrays<<<numBlocks, blockSize>>>(d_data, d_results);
    stop = std::chrono::high_resolution_clock::now();
    auto mult_time = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();

    start = std::chrono::high_resolution_clock::now();
    mod_arrays<<<numBlocks, blockSize>>>(d_data, d_results);
    stop = std::chrono::high_resolution_clock::now();
    auto mod_time = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count();
    
    auto results_size = totalThreads * sizeof(ResultsStruct);
   
    hipDeviceSynchronize();
    
    // Copy results to host
    hipMemcpy(results, d_results, results_size, hipMemcpyDeviceToHost);
    
   
    printf("Results with Thread Count: %d and Block Size: %d\n", totalThreads, blockSize);
    printf("Add Time nanoseconds:\t %ld\n", add_time);
    printf("Sub Time nanoseconds:\t %ld\n", sub_time);
    printf("Mult Time nanoseconds:\t %ld\n", mult_time);
    printf("Mod Time nanoseconds:\t %ld\n", mod_time);
   
    auto test = results[0];
    
    if ( !outputName.empty() )
    {
        print_results(outputName, totalThreads, blockSize, add_time, sub_time, mult_time, mod_time,
            data, results);
    }
}

__host__
void init_data(const int& totalThreads, const bool& pageable, MathStruct*& host_data, ResultsStruct*& host_results,
    MathStruct*& d_data, ResultsStruct*& d_results)
{
    auto data_size = totalThreads * sizeof(MathStruct);
    auto results_size = totalThreads * sizeof(ResultsStruct);
    hipMalloc((void**)&d_data, data_size);
    hipMalloc((void**)&d_results, results_size);

    if ( pageable )
    {
        host_data = (MathStruct*)malloc(data_size);
        host_results = (ResultsStruct*)malloc(results_size);
    }
    else
    {
        hipHostMalloc((void**)&host_data, data_size, hipHostMallocDefault);
        hipHostMalloc((void**)&host_results, results_size, hipHostMallocDefault);
    }

    // Used for random number generation
    std::default_random_engine generator;
    std::uniform_int_distribution<int> distribution(0,3);

    for( int i = 0; i < totalThreads; i++ )
    {
        host_data[i].a = i;
        host_data[i].b = distribution( generator );
    }
    hipMemcpy(d_data, host_data, data_size, hipMemcpyHostToDevice);
}

__host__ 
void cleanup( 
    const bool& pageable, MathStruct*& data, MathStruct*& d_data,
    ResultsStruct*& results, ResultsStruct*& d_results)
{
    hipFree(d_data);
    hipFree(d_results);
    
    if ( pageable )
    {
        free(data);
        free(results);
    }
    else
    {
        hipFree(data);
        hipFree(results);
    }
}

__host__
void execute_math_pageable_mem(
    const int& blockSize, const int& totalThreads, const int& numBlocks,
    const bool& writeResults, const std::string& outputName)
{
    MathStruct* data = nullptr;
    MathStruct* d_data = nullptr;
    ResultsStruct* results = nullptr;
    ResultsStruct* d_results = nullptr;
    init_data(totalThreads, true, data, results, d_data, d_results);
    run_kernal(blockSize, totalThreads, numBlocks, outputName, data, results,
        d_data, d_results);
    cleanup(true, data, d_data, results, d_results);
}

__host__
void execute_math_pinnable_mem(
    const int& blockSize, const int& totalThreads, const int& numBlocks,
    const bool& writeResults, const std::string& outputName)
{
    MathStruct* data = nullptr;
    MathStruct* d_data = nullptr;
    ResultsStruct* results = nullptr;
    ResultsStruct* d_results = nullptr;
    init_data(totalThreads, false, data, results, d_data, d_results);
    run_kernal(blockSize, totalThreads, numBlocks, outputName, data, results,
         d_data, d_results);
    cleanup(false, data, d_data, results, d_results);

}

int main(int argc, char** argv)
{
    // read command line arguments
    int totalThreads = 512;
    int blockSize = 256;
    bool outputResults = false;
    std::string outputName;

    if (argc >= 2) 
    {
        totalThreads = atoi(argv[1]);
    }
    if (argc >= 3)
    {
        blockSize = atoi(argv[2]);
    }
    if (argc >= 4)
    {
        outputResults = true;
        outputName = argv[3];
    }
    int numBlocks = totalThreads/blockSize;

    // validate command line arguments
    if (totalThreads % blockSize != 0)
    {
        ++numBlocks;
        totalThreads = numBlocks*blockSize;

        printf("Warning: Total thread count is not evenly divisible by the block size\n");
        printf("The total number of threads will be rounded up to %d\n", totalThreads);
    }
    execute_math_pageable_mem( blockSize, totalThreads, numBlocks, outputResults, outputName);
}
